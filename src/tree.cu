#include "hip/hip_runtime.h"
#include <gputiger/tree.hpp>

__device__
                 static tree** arena;

__device__
static int next_index;

__device__
static int arena_size;

__device__
static int* active_list;

__device__
static int active_count;

__device__
 static particle* part_begin;

struct tree_sort_type {
	array<tree*, NCHILD> tree_ptrs;
	array<range<pos_type>, NCHILD> boxes;
	array<particle*, NCHILD> begins;
	array<particle*, NCHILD> ends;
	array<monopole, NCHILD> poles;
};

__device__
void tree::initialize(particle* parts, void* data, size_t bytes) {
	int sztot = sizeof(tree) + sizeof(tree*);
	int N = (bytes - opts.nparts * sizeof(int)) / sztot;
	next_index = 0;
	arena_size = N;
	tree* ptrs = (tree*) data;
	arena = (tree**) (data + sizeof(tree) * N);
	active_list = (int*) (data + (sizeof(tree) + sizeof(tree*)) * N);
	active_count = 0;
	for (int i = 0; i < N; i++) {
		arena[i] = ptrs + i;
	}
	::part_begin = parts;
	printf("Done allocating trees\n");
}

__device__ tree* tree::alloc() {
	int index = atomicAdd(&next_index, 1);
	if (index < arena_size) {
		return arena[index];
	} else {
		printf("Out of tree memory!\n");
		__trap();
	}
}

__device__ particle* sort_parts(int* lo, int* hi, particle* swap, particle* b, particle* e, pos_type xmid, int dim) {
	const int& tid = threadIdx.x;
	const int& block_size = blockDim.x;
	if (e == b) {
		return e;
	} else {
		if (tid == 0) {
			*lo = 0;
			*hi = (e - b) - 1;
		}
		__syncthreads();
		int index;
		for (particle* part = b + tid; part < e; part += block_size) {
			if (part->x[dim] > xmid) {
				index = atomicSub(hi, 1);
			} else {
				index = atomicAdd(lo, 1);
			}
			swap[index] = *part;
		}
		__syncthreads();
		for (index = tid; index < (e - b); index += block_size) {
			b[index] = swap[index];
		}
		__syncthreads();
		return b + *lo;
	}
}

__global__
void root_tree_sort(tree* root, particle* swap_space, particle* pbegin, particle* pend, const range<pos_type> box,
		int rung) {
	__shared__ sort_workspace spaces[MAXDEPTH];
	root->sort(spaces, swap_space, pbegin, pend, box, 0, rung);
}

__global__
void tree_sort(tree_sort_type* trees, particle* swap_space, int depth, int rung) {
	__shared__ sort_workspace spaces[MAXDEPTH];
	const int& bid = blockIdx.x;
	particle* base = trees->begins[0];
	particle* b = trees->begins[bid];
	particle* e = trees->ends[bid];
	particle* this_swap = swap_space + (b - base);
	range<pos_type> box = trees->boxes[bid];
	trees->poles[bid] = trees->tree_ptrs[bid]->sort(spaces, this_swap, b, e, box, depth, rung);
	__syncthreads();
}

__device__ monopole tree::sort(sort_workspace* workspace, particle* swap_space, particle* pbegin, particle* pend,
		range<pos_type> box_, int depth_, int rung) {
	const int& tid = threadIdx.x;
	const int& block_size = blockDim.x;
	if (tid == 0) {
		part_begin = pbegin;
		part_end = pend;
		depth = depth_;
		leaf = false;
		box = box_;
	}
	pos_type midx;
	particle* mid;
	__syncthreads();
	/*if (tid == 0) {
	 for (auto* ptr = part_begin; ptr < part_end; ptr++) {
	 if (!box.in_range(ptr->x)) {
	 printf("Particle out of range at depth %i\n", depth);
	 for (int dim = 0; dim < NDIM; dim++) {
	 printf("%i %i %i\n", box.begin[dim], box.end[dim], ptr->x[dim]);
	 }
	 printf("\n");
	 __trap();
	 }
	 }
	 printf("Sorting at depth %i\n", depth);
	 }*/

	__syncthreads();
	if (pend - pbegin > opts.parts_per_bucket) {
//		auto tm = clock();
		midx = (box.begin[2] / pos_type(2) + box.end[2] / pos_type(2));
		mid = sort_parts(&workspace->lo, &workspace->hi, swap_space, pbegin, pend, midx, 2);
		if (tid == 0) {
			workspace->begin[0] = pbegin;
			workspace->end[0] = workspace->begin[4] = mid;
			workspace->end[4] = pend;
		}
		__syncthreads();
		for (int i = 0; i < 2; i++) {
			particle* b = workspace->begin[4 * i];
			particle* e = workspace->end[4 * i];
			midx = (box.begin[1] / pos_type(2) + box.end[1] / pos_type(2));
			mid = sort_parts(&workspace->lo, &workspace->hi, swap_space, b, e, midx, 1);
			if (tid == 0) {
				workspace->begin[4 * i] = b;
				workspace->end[4 * i] = workspace->begin[4 * i + 2] = mid;
				workspace->end[4 * i + 2] = e;
			}
			__syncthreads();
		}
		for (int i = 0; i < 4; i++) {
			particle* b = workspace->begin[2 * i];
			particle* e = workspace->end[2 * i];
			midx = (box.begin[0] / pos_type(2) + box.end[0] / pos_type(2));
			mid = sort_parts(&workspace->lo, &workspace->hi, swap_space, b, e, midx, 0);
			if (tid == 0) {
				workspace->begin[2 * i] = b;
				workspace->end[2 * i] = workspace->begin[2 * i + 1] = mid;
				workspace->end[2 * i + 1] = e;
			}
			__syncthreads();
		}
		__syncthreads();
		if (tid == 0) {
			for (int ci = 0; ci < NCHILD; ci++) {
				children[ci] = alloc();
			}
		}
		__syncthreads();
		box.split(workspace->cranges);
		int64_t count = 0;
		if (depth > opts.max_kernel_depth) {
			for (int ci = 0; ci < NCHILD; ci++) {
				particle* swap_base = swap_space + (workspace->begin[ci] - workspace->begin[0]);
				monopole this_pole = children[ci]->sort(workspace + 1, swap_base, workspace->begin[ci],
						workspace->end[ci], workspace->cranges[ci], depth + 1, rung);
				if (tid == 0) {
					count += this_pole.mass;
				}
				__syncthreads();
				if (tid < NDIM) {
					workspace->poles[ci][tid] = this_pole.xcom[tid];
				}
				__syncthreads();
			}
			__syncthreads();
		} else {
			__shared__ tree_sort_type* childdata;
			if (tid == 0) {
				CUDA_CHECK(hipMalloc(&childdata, sizeof(tree_sort_type)));
			}
			__syncthreads();
			if (tid < NCHILD) {
				childdata->boxes[tid] = workspace->cranges[tid];
				childdata->tree_ptrs[tid] = children[tid];
				childdata->begins[tid] = workspace->begin[tid];
				childdata->ends[tid] = workspace->end[tid];
			}
			__syncthreads();
			if (tid == 0) {
				int threadcnt;
				if (depth == opts.max_kernel_depth) {
					threadcnt = WARPSIZE;
				} else {
					threadcnt = min((int) ((part_end - part_begin) / NCHILD), (int) MAXTHREADCOUNT);
				}
				tree_sort<<<NCHILD,threadcnt>>>(childdata, swap_space, depth+1, rung);
				CUDA_CHECK(hipGetLastError());
			}
			__syncthreads();
			if (tid == 0) {
				CUDA_CHECK(hipDeviceSynchronize());
				CUDA_CHECK(hipFree(childdata));
			}
			__syncthreads();
			if (tid < NCHILD) {
				for (int dim = 0; dim < NDIM; dim++) {
					workspace->poles[tid][dim] = childdata->poles[tid].xcom[dim];
				}
				if (tid == 0) {
					count += childdata->poles[tid].mass;
				}
			}
			__syncthreads();
		}
		for (int P = NCHILD / 2; P >= 1; P /= 2) {
			for (int dim = 0; dim < NDIM; dim++) {
				if (tid < P) {
					for (int dim = 0; dim < NDIM; dim++) {
						workspace->poles[tid][dim] += workspace->poles[tid + P][dim];
					}
				}
			}
			__syncthreads();
		}
		if (tid == 0) {
			pole.mass = count;
		}
		__syncthreads();
		if (tid < NDIM) {
			pole.xcom[tid] = workspace->poles[0][tid] / count;
		}
		__syncthreads();
	} else {
		if (tid == 0) {
			leaf = true;
		}
		if (tid < WARPSIZE) {
			auto& poles = workspace->poles[tid];
			auto& count = workspace->count[tid];
			for (int dim = 0; dim < NDIM; dim++) {
				poles[dim] = 0;
			}
			count = 0;
			for (particle* p = part_begin + tid; p < part_end; p += block_size) {
				for (int dim = 0; dim < NDIM; dim++) {
					poles[dim] += p->x[dim];
				}
				if (p->rung >= rung) {
					int index = atomicAdd(&active_count, 1);
					active_list[index] = (p - part_begin);
				}
			}
			for (int P = block_size / 2; P >= 1; P /= 2) {
				if (tid < P) {
					for (int dim = 0; dim < NDIM; dim++) {
						poles[dim] += workspace->poles[tid + P][dim];
					}
				}
				__syncthreads();
			}
			if (tid == 0) {
				pole.mass = count;
			}
			__syncthreads();
			if (tid < NDIM) {
				pole.xcom[tid] = pos_type(poles[tid] / pole.mass);
			}
			__syncthreads();
		}
		__syncthreads();
	}
	if (tid == 0) {
		pole.radius = (box.end[0] - box.begin[0]) / 2;
	}
	__syncthreads();
	return pole;
}
