#include "hip/hip_runtime.h"
class cmplx {
	float x, y;
public:
	__device__ cmplx() = default;
	__device__ cmplx(float a) {
		x = a;
		y = 0.f;
	}
	__device__ cmplx(float a, float b) {
		x = a;
		y = b;
	}
	__device__ cmplx& operator+=(cmplx other) {
		x += other.x;
		y += other.y;
		return *this;
	}
	__device__ cmplx& operator-=(cmplx other) {
		x -= other.x;
		y -= other.y;
		return *this;
	}
	__device__ cmplx operator*(cmplx other) const {
		cmplx a;
		a.x = x * other.x - y * other.y;
		a.y = x * other.y + y * other.x;
		return a;
	}
	__device__ cmplx operator/(cmplx other) const {
		return *this * other.conj() / other.norm();
	}
	__device__ cmplx operator/(float other) const {
		cmplx b;
		b.x = x / other;
		b.y = y / other;
		return b;
	}
	__device__ cmplx operator*(float other) const {
		cmplx b;
		b.x = x * other;
		b.y = y * other;
		return b;
	}
	__device__ cmplx operator+(cmplx other) const {
		cmplx a;
		a.x = x + other.x;
		a.y = y + other.y;
		return a;
	}
	__device__ cmplx operator-(cmplx other) const {
		cmplx a;
		a.x = x - other.x;
		a.y = y - other.y;
		return a;
	}
	__device__ cmplx conj() const {
		cmplx a;
		a.x = x;
		a.y = -y;
		return a;
	}
	__device__
	float real() const {
		return x;
	}
	__device__
	float imag() const {
		return y;
	}
	__device__
	float norm() const {
		return ((*this) * conj()).real();
	}
	__device__
	float abs() const {
		return sqrtf(norm());
	}
	__device__ cmplx operator-() const {
		cmplx a;
		a.x = -x;
		a.y = -y;
		return a;
	}
};

__device__ cmplx operator*(float a, cmplx b) {
	return b * a;
}

#include <nvfunctional>
#include <gputiger/chemistry.hpp>
#include <gputiger/options.hpp>
#include <gputiger/constants.hpp>
#include <gputiger/util.hpp>
#include <gputiger/zero_order.hpp>
#include <gputiger/boltzmann.hpp>
#include <gputiger/zeldovich.hpp>
#include <gputiger/particle.hpp>

#define BLOCK_SIZE 256

__device__ zero_order_universe *zeroverse_ptr;

__device__ float test(float x) {
	return x * x;
}
__global__
void main_kernel(void* arena, particle* host_parts, cosmic_parameters opts) {
	const int thread = threadIdx.x;
	const int block_size = blockDim.x;
	__shared__
	float *result_ptr;
	__shared__ sigma8_integrand *func_ptr;
	__shared__ interp_functor<float>* den_k;
	__shared__ interp_functor<float>* vel_k;
	__shared__ cos_state* states;
	__shared__ cmplx* basis;
	const int N = opts.Ngrid;
	const int N3 = N * N * N;
	cmplx* phi = (cmplx*) arena;
	cmplx* rands = ((cmplx*) arena) + N3;
	if (thread == 0) {
		zeroverse_ptr = new zero_order_universe;
		create_zero_order_universe(zeroverse_ptr, opts, 1.0);
	}
	__syncthreads();

	float kmin = 2.0 * (float) M_PI / opts.box_size;
	float kmax = kmin * (float) (opts.Ngrid / 2);
	kmax *= SQRT(3);
	int Nk = opts.Ngrid * SQRT(3) + 1;
	if (thread == 0) {
		printf("\nNormalizing Einstein Boltzman solutsions to a present day sigma8 of %e\n", opts.sigma8);
		result_ptr = new float;
		func_ptr = new sigma8_integrand;
		CUDA_CHECK(hipMalloc(&states, sizeof(cos_state) * Nk));
		den_k = new interp_functor<float>;
		vel_k = new interp_functor<float>;
		basis = new cmplx[opts.Ngrid / 2];
		func_ptr->uni = zeroverse_ptr;
		func_ptr->littleh = opts.h;
		integrate<sigma8_integrand, float> <<<1, BLOCK_SIZE>>>(func_ptr,
				(float) LOG(0.25 / 32.0 * opts.h), (float) LOG(0.25 * 32.0 * opts.h), result_ptr, (float) 1.0e-6);
		CUDA_CHECK(hipGetLastError());
		CUDA_CHECK(hipDeviceSynchronize());
		*result_ptr = SQRT(opts.sigma8 * opts.sigma8 / *result_ptr);
		printf("The normalization value is %e\n", *result_ptr);
	}
	__syncthreads();
	if (thread == 0) {
		printf("Computing start time for non-linear evolution\n");
	}
	float normalization = *result_ptr;
	if (thread == 0) {
		printf("wave number range %e to %e Mpc^-1 for %i^3 grid and box size of %e Mpc\n", kmin, kmax, opts.Ngrid,
				opts.box_size);
	}
	fft_basis(basis, opts.Ngrid);
	generate_random_normals(rands, N * N * N);
	__syncthreads();
	einstein_boltzmann_init_set(states, zeroverse_ptr, kmin, kmax, Nk, zeroverse_ptr->amin, normalization);
	int iter = 0;
	float dloga = 2.f;
	float logamin = LOG(zeroverse_ptr->amin);
	float logamax = LOG(zeroverse_ptr->amax);
	float loga = logamin;
	float drho;
	float a;
	for (; loga < LOG(zeroverse_ptr->amax) - dloga; loga += dloga) {
		if (iter > 1) {
			dloga = 2*(opts.max_overden - drho) / opts.max_overden;
		}
		float amin = EXP(loga);
		float amax = EXP(loga + dloga);
		if (thread == 0) {
			printf("Advancing Einstein Boltzmann solutions from redshift %.1f to %.1f\n", 1 / amin - 1, 1 / amax - 1);
		}
		einstein_boltzmann_interpolation_function(den_k, vel_k, states, zeroverse_ptr, kmin, kmax, Nk, amin, amax);
		__syncthreads();
		drho = zeldovich_overdensity(phi, basis, rands, *den_k, opts.box_size, opts.Ngrid);
		__syncthreads();
		if (thread == 0) {
			printf("Maximum over/under density is %e\n", drho);
		}
		__syncthreads();
		if (iter > 0) {
			if (drho > opts.max_overden) {
				a = amax;
				break;
			}
		}
		__syncthreads();
		iter++;
	}
	__syncthreads();
	generate_random_normals(rands, N * N * N);
	__syncthreads();
	for (int dim = 0; dim < NDIM; dim++) {
		float vmax = zeldovich_velocities(phi, basis, rands, *vel_k, opts.box_size, opts.Ngrid, 0);
		__syncthreads();
		for (int ij = thread; ij < N*N; ij += block_size) {
			int i = ij / N;
			int j = ij % N;
			for (int k = 0; k < N; k++) {
				const int l = N * (N * i + j) + k;
				float v = phi[l].real();
				host_parts[l].v[0] = v;
			}
		}
		__syncthreads();
		float xdisp = zeldovich_displacements(phi, basis, rands, *den_k, opts.box_size, opts.Ngrid, 0);
		__syncthreads();
		for (int ij = thread; ij < N*N; ij += block_size) {
			int i = ij / N;
			int j = ij % N;
			for (int k = 0; k < N; k++) {
				const int l = N * (N * i + j) + k;
				const int I[NDIM] = { i, j, k };
				float x = ((float) I[dim] + 0.5f) / (float) N - 0.5f;
				x += phi[l].real();
				float test1 = host_parts[l].v[0] / phi[l].real();
				printf("%e\n", test1);
				host_parts[l].x[dim] = float_to_pos(x);
			}
		}
		__syncthreads();
	}
	__syncthreads();
	for (int ij = thread; ij < N*N; ij += block_size) {
		int i = ij / N;
		int j = ij % N;
		for (int k = 0; k < N; k++) {
			const int l = N * (N * i + j) + k;
			host_parts[l].rung = 0;
		}
	}
	__syncthreads();
	if (thread == 0) {
		delete zeroverse_ptr;
		delete result_ptr;
		delete func_ptr;
		CUDA_CHECK(hipFree(states));
		delete den_k;
		delete vel_k;
		delete[] basis;
	}
}

int main() {

	cosmic_parameters params;

	size_t stack_size;
	size_t desired_stack_size = 4 * 1024;
	CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, desired_stack_size));
	CUDA_CHECK(hipDeviceGetLimit(&stack_size, hipLimitStackSize));
//	CUDA_CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
	particle* parts_ptr;
	printf("Stack Size = %li\n", stack_size);
	params.h = 0.7;
	params.Neff = 3.086;
	params.Y = 0.24;
	params.omega_b = 0.05;
	params.omega_c = 0.25;
	params.Theta = 1.0;
	params.Ngrid = 256;
	params.sigma8 = 0.8367;
	params.max_overden = 0.9;
	params.box_size = 100.0;
	double omega_r = 32.0 * M_PI / 3.0 * constants::G * constants::sigma
			* (1 + params.Neff * (7. / 8.0) * std::pow(4. / 11., 4. / 3.)) * std::pow(constants::H0, -2)
			* std::pow(constants::c, -3) * std::pow(2.73 * params.Theta, 4) * std::pow(params.h, -2);
	params.omega_nu = omega_r * params.Neff / (8.0 / 7.0 * std::pow(11.0 / 4.0, 4.0 / 3.0) + params.Neff);
	params.omega_gam = omega_r - params.omega_nu;
	void* arena;
	const int N = params.Ngrid;
	const int N3 = N * N * N;
	particle* parts_device;
	CUDA_CHECK(hipHostAlloc(&parts_ptr, sizeof(particle) * N3, hipHostMallocMapped));
	CUDA_CHECK(hipHostGetDevicePointer(&parts_device, parts_ptr, 0));
	CUDA_CHECK(hipMalloc(&arena, 2 * sizeof(cmplx) * N3));

	main_kernel<<<1, BLOCK_SIZE>>>(arena, parts_device, params);
	CUDA_CHECK(hipGetLastError());

	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipFree(arena));
	CUDA_CHECK(hipHostFree(parts_ptr));
}
